#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


// includes, system
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <chrono>

// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

// Complex data type
typedef float2 Complex;

#define SIGNAL_SIZE 300000000
#define REPEAT (32)

int main(int argc, char **argv) { 
  findCudaDevice(argc, (const char **)argv); 

  // Allocate host memory for the signal
  Complex *h_signal = reinterpret_cast<Complex *>(malloc(sizeof(Complex) * SIGNAL_SIZE));
  // Initialize the memory for the signal
  for (unsigned long long int i = 0; i < SIGNAL_SIZE; ++i) {
    h_signal[i].x = rand() / static_cast<float>(RAND_MAX);
    h_signal[i].y = 0;
  }

  unsigned long long int mem_size = sizeof(Complex) * SIGNAL_SIZE;

  // Allocate device memory for signal
  Complex *d_signal;
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_signal), mem_size));
  // Copy host memory to device
  checkCudaErrors(hipMemcpy(d_signal, h_signal, mem_size, hipMemcpyHostToDevice));

  // CUFFT plan simple API
  hipfftHandle plan;
  checkCudaErrors(hipfftPlan1d(&plan, SIGNAL_SIZE, HIPFFT_C2C, 1));



  for (int i = 0; i < 2; i++) hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(d_signal), reinterpret_cast<hipfftComplex *>(d_signal), HIPFFT_FORWARD);
  checkCudaErrors(hipDeviceSynchronize());

  // printf("Transforming signal hipfftExecC2C\n");

  uint64_t start_ts, end_ts;
  start_ts = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now().time_since_epoch()).count();

  for (int i = 0; i < REPEAT; i++) hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(d_signal), reinterpret_cast<hipfftComplex *>(d_signal), HIPFFT_FORWARD);
  checkCudaErrors(hipDeviceSynchronize());

  end_ts = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now().time_since_epoch()).count();



  // printf("Done\n");
  // printf("Kernel Execution Time: %f ms\n", (end_ts - start_ts) / 1000.0 / REPEAT);
  // printf("Total runtime: %f ms\n", (end_ts - start_ts) / 1000.0);

  // Write the timestamps to a file
  std::ofstream outfile;
  outfile.open("timestamps.csv");
  outfile << "timestamp" << std::endl;
  outfile << start_ts << std::endl;
  outfile << end_ts << std::endl;
  outfile.close();

  // Copy device memory to host
  // Complex *h_fft_signal = h_signal;
  // checkCudaErrors(hipMemcpy(h_fft_signal, d_signal, mem_size, hipMemcpyDeviceToHost));

  // Destroy CUFFT context
  checkCudaErrors(hipfftDestroy(plan));

  // cleanup memory
  free(h_signal);
  checkCudaErrors(hipFree(d_signal));

  exit(EXIT_SUCCESS);
}