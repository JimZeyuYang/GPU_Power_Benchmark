#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


// includes, system
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>

// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

// Complex data type
typedef float2 Complex;

// The filter size is assumed to be a number smaller than the signal size
#define SIGNAL_SIZE 200000000

int main(int argc, char **argv) { 
  findCudaDevice(argc, (const char **)argv); 

  // Allocate host memory for the signal
  Complex *h_signal = reinterpret_cast<Complex *>(malloc(sizeof(Complex) * SIGNAL_SIZE));
  // Initialize the memory for the signal
  for (unsigned long long int i = 0; i < SIGNAL_SIZE; ++i) {
    h_signal[i].x = rand() / static_cast<float>(RAND_MAX);
    h_signal[i].y = 0;
  }

  unsigned long long int mem_size = sizeof(Complex) * SIGNAL_SIZE;

  // Allocate device memory for signal
  Complex *d_signal;
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_signal), mem_size));
  // Copy host memory to device
  checkCudaErrors(hipMemcpy(d_signal, h_signal, mem_size, hipMemcpyHostToDevice));

  // CUFFT plan simple API
  hipfftHandle plan;
  checkCudaErrors(hipfftPlan1d(&plan, SIGNAL_SIZE, HIPFFT_C2C, 1));

  printf("Transforming signal hipfftExecC2C\n");
  for (int i = 0; i < 100; i++) {
    // Transform signal and kernel
    checkCudaErrors(hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(d_signal),
                                reinterpret_cast<hipfftComplex *>(d_signal),
                                HIPFFT_FORWARD));
  }
  // sychronize
  checkCudaErrors(hipDeviceSynchronize());
  printf("Done\n");

  // Copy device memory to host
  Complex *h_fft_signal = h_signal;
  checkCudaErrors(hipMemcpy(h_fft_signal, d_signal, mem_size, hipMemcpyDeviceToHost));

  // Destroy CUFFT context
  checkCudaErrors(hipfftDestroy(plan));

  // cleanup memory
  free(h_signal);
  checkCudaErrors(hipFree(d_signal));

  exit(EXIT_SUCCESS);
}