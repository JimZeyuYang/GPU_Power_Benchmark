#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <chrono>
#include <unistd.h>
#include <fstream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define CHECK 1.0

__global__ void my_first_kernel(float *x, int niter) {
    int tid = threadIdx.x + blockDim.x*blockIdx.x;

    #pragma unroll
    for (int i=0; i<niter; i++) {
        x[tid] *= 2;
        x[tid] += 2;
        x[tid] /= 2;
        x[tid] -= 1;
    }
}

hipDeviceProp_t getDeviceProperties() {
    int devCount;
    hipGetDeviceCount(&devCount);
    if (devCount == 0) {
        printf("No CUDA devices found\n");
    }

    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);

    return devProp;
}

void printDevProps(hipDeviceProp_t devProp) {
    std::cout << "Device name: " << devProp.name << std::endl;
    std::cout << "Number of SMs: " << devProp.multiProcessorCount << std::endl;
    std::cout << "Maximum threads per block: " << devProp.maxThreadsPerBlock << std::endl;    
    std::cout << "Maximum blocks per grid: " << devProp.maxGridSize[0] << std::endl;
    std::cout << "Maximum threads per SM: " << devProp.maxThreadsPerMultiProcessor << std::endl;
    std::cout << "Maximum shared memory per block: " << devProp.sharedMemPerBlock << " B" << std::endl;
    std::cout << "Maximum shared memory per SM: " << devProp.sharedMemPerMultiprocessor << " B" << std::endl;
    std::cout << "Maximum global memory: " << devProp.totalGlobalMem << " B" << std::endl;
    std::cout << "Maximum blocks per SM: " << devProp.maxBlocksPerMultiProcessor << std::endl;
    std::cout << "Maximum threads per SM: " << devProp.maxThreadsPerMultiProcessor << std::endl;
}

int main(int argc, const char **argv) {
    int delay = std::stoi(argv[1]);
    int niter = std::stoi(argv[2]);
    int testLength = std::stoi(argv[3]);
    std::string result_dir = argv[4];
    

    int dev = findCudaDevice(argc, (const char **) argv);
    if (dev == -1) return EXIT_FAILURE;
    hipDeviceProp_t devProp = getDeviceProperties();
    // printDevProps(devProp);

    int nblocks  = devProp.maxBlocksPerMultiProcessor * devProp.multiProcessorCount;
    int nthreads = devProp.maxThreadsPerBlock;

    int nsize    = nblocks * nthreads;
    // std::cout << "nblocks: " << nblocks << std::endl;
    // std::cout << "nthreads: " << nthreads << std::endl;
    // std::cout << "nsize: " << nsize << std::endl;

    float *h_x, *d_x;
    h_x = (float *)malloc(nsize*sizeof(float));
    checkCudaErrors(hipMalloc((void **)&d_x, nsize*sizeof(float)));

    // populate host array with 1
    for (int i=0; i<nsize; i++) h_x[i] = CHECK;

    // copy host array to device
    checkCudaErrors(hipMemcpy(d_x,h_x,nsize*sizeof(float), hipMemcpyHostToDevice));
    
    // Measurement begins
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    uint64_t timestamps[2*testLength+1];
    sleep(1);
    
    for (int i=0; i<testLength; i++) {
        timestamps[2*i] = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
        
        // hipEventRecord(start); 

        my_first_kernel<<<nblocks,nthreads>>>(d_x, niter);
        getLastCudaError("my_first_kernel execution failed\n");
        
        // hipEventRecord(stop);
        // hipEventSynchronize(stop);
        hipDeviceSynchronize();

        // float milliseconds = 0;
        // hipEventElapsedTime(&milliseconds, start, stop);
        // std::cout << "Elapsed time:    " << milliseconds << " ms" << std::endl;
        
        // Record the end time
        timestamps[2*i+1] = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
        usleep(delay*1000);
        // std::cout << "Sleeping for:    " << delay << "      ms" << std::endl;
    }
    timestamps[2*testLength] = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
    sleep(1);
    
    // Write the timestamps to a file
    std::ofstream outfile;
    std::string filename = result_dir + "/timestamps.csv";
    outfile.open(filename);
    outfile << "timestamp" << std::endl;
    for (int i = 0; i < 2*testLength+1; i++) {
        outfile << timestamps[i] << std::endl;
        outfile << timestamps[i] << std::endl;
    }
    outfile.close();

    checkCudaErrors(hipMemcpy(h_x,d_x,nsize*sizeof(float), hipMemcpyDeviceToHost));

    // Check if the result is correct
    float sum = 0.0;
    for (int i=0; i<nsize; i++) sum += h_x[i];

    // raise a error if sum/nsize != CHECK
    if (sum/nsize != CHECK) {
        printf("Error: result is %f instead of %f\n", sum/nsize, CHECK);
        exit(EXIT_FAILURE);
    }
    

    // free memory 
    checkCudaErrors(hipFree(d_x));
    free(h_x);

    // CUDA exit -- needed to flush printf write buffer
    hipDeviceReset();

    return 0;
}
